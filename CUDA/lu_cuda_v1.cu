#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define N 1024
#define BLOCK_SIZE 32

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}


static void init_array(int n, float *A)
{    
    int i, j;
    
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++)
            A[i*n+j] = ((float)(i + 1) * (j + 1)) / n;
    }
}

static void print_array(int n, float *A)
{
    int i, j;
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%f ", A[i*n+j]);
        }
        printf("\n");
    }
    printf("\n");
}


static void kernel_lu(int n, float *A)
{
    int i, j, k;

    for (k = 0; k < n; k++)
    {
        for (j = k + 1; j < n; j++)
            A[k*n+j] = A[k*n+j] / A[k*n+k];
        for (i = k + 1; i < n; i++)
            for (j = k + 1; j < n; j++)
                A[i*n+j] = A[i*n+j] + A[i*n+k] * A[k*n+j];
    }
}

__global__ void gpu_kernel_lu(float * __restrict__ A, int k, int n)
{
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (i<n && j<n && i>k && j>k)
    {
	if(i==n-1)
	{
            A[k*n+j] = A[k*n+j] / A[k*n+k];
	}
        __syncthreads();

        A[i*n+j] = A[i*n+j] + A[i*n+k] * A[k*n+j];
    }
}

int main(int argc, char **argv)
{
    int n = N;
    int k = 0;
    struct timespec rt[2];
    double wt;
    float *A;
    A = (float *)malloc(n * n * sizeof(*A));

    init_array(n, A);
    clock_gettime(CLOCK_REALTIME, rt + 0);
    kernel_lu(n, A);
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("KERNEL_LU (Host) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * n * n * n / (1.0e9 * wt));
    //print_array(n, A);

    init_array(n, A);
    
    //cudaMalloc
    float *d_A;
    gpuErrchk(hipMalloc((void **)&d_A, sizeof(float) * n * n));

    //cudamemcopy
    struct timespec rt2[2];
    double wt2;
    clock_gettime(CLOCK_REALTIME, rt2 + 0);

    gpuErrchk(hipMemcpy(d_A, A, sizeof(float) * n * n, hipMemcpyHostToDevice));    

    dim3 dimGrid((n+BLOCK_SIZE-1)/BLOCK_SIZE, (n+BLOCK_SIZE-1)/BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
   
   
    for (k = 0; k<n; k++)
        gpu_kernel_lu<<<dimGrid, dimBlock>>>(d_A, k, n);

   
    gpuErrchk(hipMemcpy(A, d_A, sizeof(float) * n * n, hipMemcpyDeviceToHost));    

    clock_gettime(CLOCK_REALTIME, rt2 + 1);

    wt2 = (rt2[1].tv_sec - rt2[0].tv_sec) + 1.0e-9 * (rt2[1].tv_nsec - rt2[0].tv_nsec);
    printf("KERNEL_LU (GPU) : %9.3f sec %9.1f GFLOPS\n", wt2, 2.0 * n * n * n / (1.0e9 * wt2));

    //print_array(n, A);

    hipDeviceReset();
    return 0;
}
