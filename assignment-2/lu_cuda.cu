
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>

#define N 12
//double A[N][N];

static void init_array(int n, double **A)
{
    
    //printf("Entro in init! \n");
    
    int i, j;
    
    for (i = 0; i < n; i++) {
        //printf("%d \n", i);
        for (j = 0; j < n; j++)
            //printf("%d \n", j);
            A[i][j] = ((double)(i + 1) * (j + 1)) / n;
    }
}

static void print_array(int n, double **A)
{
    int i, j;
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            printf("%f ", A[i][j]);
        }
        printf("\n");
    }
    printf("\n");
}

static void kernel_lu(int n, double **A)
{
    int i, j, k;

    for (k = 0; k < n; k++)
    {
        for (j = k + 1; j < n; j++)
            A[k][j] = A[k][j] / A[k][k];
        for (i = k + 1; i < n; i++)
            for (j = k + 1; j < n; j++)
                A[i][j] = A[i][j] + A[i][k] * A[k][j];
    }
}

int main(int argc, char **argv)
{
    int n = N;
    struct timespec rt[2];
    double wt;
    double **A;
    A = (double **)malloc(n * sizeof(*A));

    for (int i = 0; i<n; i++){
        A[i] = (double *)malloc(n * sizeof(A));
    }
    //A = (double *) malloc(sizeof(*A) * n * n);

    init_array(n, A);
    print_array(n, A);
    clock_gettime(CLOCK_REALTIME, rt + 0);
    kernel_lu(n, A);
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("KERNEL_LU (Host) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * n * n * n / (1.0e9 * wt));
    print_array(n, A);
    
    //kernel_lu(n);
    
}
